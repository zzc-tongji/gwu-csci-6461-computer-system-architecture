#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "calculate.h"
#include "cuda_helper.h"
#include "matrix.h"

__global__ void KernelDotProduct(int *in_1, int *in_2, int size, int *out)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_size = size * size;
  if (index >= total_size)
  {
    return;
  }
  int row = index / size;
  int column = index % size;
  int result = 0;
  for (int i = 0; i < size; i++)
  {
    result += in_1[row * size + i] * in_2[i * size + column];
  }
  out[index] = result;
}

void DotProductByGpu(Matrix *input_1, Matrix *input_2, Matrix *output)
{
  int size = output->size;
  int total_size = output->total_size;
  CudaErrorHandler(hipSetDevice(0));
  // allocate
  int *in_1 = NULL;
  int *in_2 = NULL;
  int *out = NULL;
  CudaErrorHandler(hipMalloc(&in_1, total_size * sizeof(int)));
  CudaErrorHandler(hipMalloc(&in_2, total_size * sizeof(int)));
  CudaErrorHandler(hipMalloc(&out, total_size * sizeof(int)));
  // host => device
  CudaErrorHandler(hipMemcpy(in_1, input_1->data, total_size * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice));
  CudaErrorHandler(hipMemcpy(in_2, input_2->data, total_size * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice));
  // Launch a kernel on the GPU with one thread for each element.
  KernelDotProduct<<<total_size / 1024 + 1, 1024>>>(in_1, in_2, size, out);
  // Check for any errors launching the kernel
  CudaErrorHandler(hipGetLastError());
  // host <= device
  CudaErrorHandler(hipMemcpy(output->data, out, total_size * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost));
  // synchronize
  CudaErrorHandler(hipDeviceSynchronize());
}
